#include "hip/hip_runtime.h"
#include <stdio.h>

const int BLOCK_SIZE = 32;

__global__ void fill_1_block(float *array) {
  int idx = threadIdx.x;
  array[idx] = 1;
}

__global__ void fill_0_block(float *array) {
  int idx = threadIdx.x;
  array[idx] = 0;
}

float * fill_random_block(float *array, int n) {
  for (int i = 0; i < n; ++i) {
    array[i] = (float)rand()/(float)(RAND_MAX/1024);
  }
  return array;
}

float * generate_filter(float *array, int n) {
  for (int i = 0; i < n; ++i) {
    array[i] = (i % 2);
  }
  return array;
}

__global__ void sum_array(float * array, float * result_array, int n) {
  int idx = threadIdx.x;
  if (idx < n) {
    atomicAdd(result_array, array[idx]);
  }
}


void task1() {
  int n = 1 << 20;
  printf("%d\n", n);
  float * device_array, * device_result_array;
  float * array = (float*)malloc(n*sizeof(float));
  float * result_array = (float*)malloc(n*sizeof(float));

  hipMalloc(&device_array, n*sizeof(float));
  hipMalloc(&device_result_array, n*sizeof(float));

  hipMemcpy(device_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_result_array, result_array, n*sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 1024;
  int gridSize = (int)ceil((float)n/blockSize);

  fill_1_block<<<gridSize, blockSize>>>(device_array);
  fill_0_block<<<gridSize, blockSize>>>(device_result_array);

  sum_array<<<gridSize, blockSize>>>(device_array, device_result_array, n);
  hipMemcpy(result_array, device_result_array, n*sizeof(float), hipMemcpyDeviceToHost);

  printf("Sum: %f\n", result_array[0]);

  hipFree(device_array);
  hipFree(device_result_array);
  free(array);
  free(result_array);
}

__global__ void get_min_array(float *array, float *min_results) {
  extern __shared__ float mintile[BLOCK_SIZE];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  mintile[tid] = array[i];
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      if (mintile[tid + s] < mintile[tid]) {
        mintile[tid] = mintile[tid + s];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    min_results[blockIdx.x] = mintile[0];
  }
}

__global__ void get_final_min_array(float * min_results) {
  __shared__ float mintile[BLOCK_SIZE];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  mintile[tid] = min_results[i];
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      if (mintile[tid + s] < mintile[tid]) {
        mintile[tid] = mintile[tid + s];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    min_results[blockIdx.x] = mintile[0];
  }
}

void task2() {
  int n = 1 << 10;
  printf("%d\n", n);
  float * device_array, * device_result_array;
  float * array = (float*)malloc(n*sizeof(float));
  float * result_array = (float*)malloc(n*sizeof(float));

  array = fill_random_block(array, n);
  float min_value = 1e9;
  for (int i = 0; i < n; ++i) {
    printf("%f\t", array[i]);
    min_value = min(min_value, array[i]);
  }
  printf("\nMin value: %f\n", min_value);

  hipMalloc(&device_array, n*sizeof(float));
  hipMalloc(&device_result_array, n*sizeof(float));

  hipMemcpy(device_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_result_array, result_array, n*sizeof(float), hipMemcpyHostToDevice);

  int blockSize = BLOCK_SIZE;
  int gridSize = (int)ceil((float)n/blockSize);

  fill_0_block<<<gridSize, blockSize>>>(device_result_array);

  get_min_array<<<gridSize, blockSize>>>(device_array, device_result_array);
  get_final_min_array<<<1, blockSize>>>(device_result_array);

  hipMemcpy(result_array, device_result_array, n*sizeof(float), hipMemcpyDeviceToHost);

  printf("Min value: %f\n", result_array[0]);
  for (int i = 0; i < blockSize; ++i) {
    printf("%f\t", result_array[i]);
  }

  hipFree(device_array);
  hipFree(device_result_array);
  free(array);
  free(result_array);
}

__global__ void convolute(float * array, float * filter, float * result_array, int array_size, int filter_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float value = 0;

  int start_point =  idx - (filter_size / 2);
  for (int i = 0; i < filter_size; ++i) {
    int current_position = start_point + i;
    if (current_position < 0 || current_position >= array_size) {
      continue;
    }
    value = value + array[current_position] * filter[i];
  }
  result_array[idx] = value;
}

void task3() {
  int n = 1 << 10;
  int m = 32;
  printf("%d\n", n);
  float * device_array, * device_result_array, * device_filter;

  float * array = (float*)malloc(n*sizeof(float));
  float * result_array = (float*)malloc(n*sizeof(float));
  float * filter = (float*)malloc(m*sizeof(float));

  array = fill_random_block(array, n);
  filter = generate_filter(filter, m);

  hipMalloc(&device_array, n*sizeof(float));
  hipMalloc(&device_result_array, n*sizeof(float));
  hipMalloc(&device_filter, m*sizeof(float));

  hipMemcpy(device_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_result_array, result_array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_filter, filter, m*sizeof(float), hipMemcpyHostToDevice);


  int blockSize = BLOCK_SIZE;
  int gridSize = (int)ceil((float)n/blockSize);

  fill_0_block<<<gridSize, blockSize>>>(device_result_array);

  convolute<<<gridSize, blockSize>>>(device_array, device_filter, device_result_array, n, m);

  hipMemcpy(result_array, device_result_array, n*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < n; ++i) {
    printf("%f\t", result_array[i]);
  }

  hipFree(device_array);
  hipFree(device_result_array);
  hipFree(device_filter);
  free(array);
  free(result_array);
  free(filter);
}

int main() {
//  task1();
//  task2();
  task3();
  return 0;
}
